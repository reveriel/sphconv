#include "hip/hip_runtime.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/numeric_types.h"

#include "cutlass/util/host_tensor.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>


inline char const* to_string(cutlass::Status status)
{

    switch (status) {
    case cutlass::Status::kSuccess:
        return "kSuccess";
    case cutlass::Status::kErrorMisalignedOperand:
        return "kErrorMisalignedOperand";
    case cutlass::Status::kErrorInvalidLayout:
        return "kErrorInvalidLayout";
    case cutlass::Status::kErrorInvalidProblem:
        return "kErrorInvalidProblem";
    case cutlass::Status::kErrorNotSupported:
        return "kErrorNotSupported";
    case cutlass::Status::kErrorWorkspaceNull:
        return "kErrorWorkspaceNull";
    case cutlass::Status::kErrorInternal:
        return "kErrorInternal";
    case cutlass::Status::kInvalid:
        return "kInvalid";
    default:
        break;
    }
    return "invalid";
}

template <typename Gemm>
class Test
{
    using ElementCompute = typename Gemm::EpilogueOutputOp::ElementCompute;

public:
    typename Gemm::TensorRefA ref_A;
    typename Gemm::TensorRefB ref_B;
    typename Gemm::TensorRefC ref_C;
    typename Gemm::TensorRefD ref_D;

    Test(torch::Tensor A,
         torch::Tensor B,
         torch::Tensor C)
    {
        auto a = A.packed_accessor64<float, 2, torch::RestrictPtrTraits>();
        auto b = B.packed_accessor64<float, 2, torch::RestrictPtrTraits>();
        auto c = C.packed_accessor64<float, 2, torch::RestrictPtrTraits>();

        ref_A = typename Gemm::TensorRefA((float*)A.data_ptr(), cutlass::layout::RowMajor(a.stride(0)));
        ref_B = typename Gemm::TensorRefB((float*)B.data_ptr(), cutlass::layout::RowMajor(b.stride(0)));
        ref_C = typename Gemm::TensorRefC((float*)C.data_ptr(), cutlass::layout::RowMajor(c.stride(0)));
        ref_D = typename Gemm::TensorRefD((float*)C.data_ptr(), cutlass::layout::RowMajor(c.stride(0)));
    }

    bool run(
        cutlass::gemm::GemmCoord problem_size,
        int split_k_slices = 1,
        ElementCompute alpha = ElementCompute(1),
        ElementCompute beta = ElementCompute(0))
    {

        // D = A * B + C
        typename Gemm::Arguments arguments{
            problem_size,
            ref_A,
            ref_B,
            ref_C,
            ref_D,
            {alpha, beta},
            split_k_slices};

        Gemm gemm_op;

        size_t workspace_size = Gemm::get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
        cutlass::Status status = gemm_op.initialize(arguments, workspace.get());

        if (status != cutlass::Status::kSuccess) {
            hipError_t error = hipGetLastError();
            std::cerr << "This test is not supported: " << hipGetErrorString(error) << "\n";
            return true;
        }

        //
        // Run the GEMM
        //

        status = gemm_op();

        std::cout << to_string(status) << std::endl;
        // EXPECT_TRUE(status == cutlass::Status::kSuccess) << to_string(status);
        return status == cutlass::Status::kSuccess;
    }
};

template <int M, int N, int K>
void test_cutlass(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor C)
{
    using precision = float;
    using ThreadblockShape = cutlass::gemm::GemmShape<8, 32, 8>;
    using WarpShape = cutlass::gemm::GemmShape<8, 32, 8>;

    static int const kEpilogueElementsPerAccess = 1;
    using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;
    using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombination<
        precision, kEpilogueElementsPerAccess, precision, precision>;

    using Gemm = cutlass::gemm::device::Gemm<
        precision, cutlass::layout::RowMajor,
        precision, cutlass::layout::RowMajor,
        precision, cutlass::layout::RowMajor,
        precision,
        cutlass::arch::OpClassSimt,
        cutlass::arch::Sm50,
        ThreadblockShape, WarpShape, InstructionShape,
        EpilogueOutputOp,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        2 // Stages
        >;

    cutlass::gemm::GemmCoord problem_size(M, N, K);

    Test<Gemm> test(A, B, C);
    double alpha = 1;
    double beta = 1;
    int split_k = 1;

    using ElementCompute = typename Gemm::EpilogueOutputOp::ElementCompute;

    //  D = alpha * A * B + beta * C
    test.run(problem_size, split_k,
             cutlass::from_real<ElementCompute>(alpha),
             cutlass::from_real<ElementCompute>(beta));
}

int main()
{
    const int M = 8;
    const int N = 32;
    const int K = 64;

    torch::manual_seed(0);
    auto options = torch::TensorOptions().dtype(torch::kFloat32).layout(torch::kStrided).device(torch::kCUDA, 0).requires_grad(false);
    // torch::Tensor A = torch::randint(40, {M, K}, options);
    // torch::Tensor B = torch::randint(40, {K, N}, options);
    torch::Tensor A = torch::randn({M, K}, options);
    torch::Tensor B = torch::randn({K, N}, options);
    torch::Tensor C_torch = torch::zeros({M, N}, options);
    torch::mm_out(C_torch, A, B);

    torch::Tensor C_cutlass = torch::zeros({M, N}, options);
    test_cutlass<M, N, K>(A, B, C_cutlass);


    std::cout << " C_torch = \n" << C_torch << std::endl;
    std::cout << " cutlass result C2 = \n" << C_cutlass << std::endl;

    std::cout << "distance = " << (C_cutlass - C_torch).abs().sum() << std::endl;

    return 0;
}