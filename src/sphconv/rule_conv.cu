#include "cutlass/gemm/gemm.h"
#include "debug_utils.h"
#include "default_conv.cu.h"
#include "rule_conv_kernel.cu.h"
#include "threadblock_swizzle.h"
#include "timer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>

namespace sphconv
{

template <typename T, int N>
using GpuTensor = torch::PackedTensorAccessor32<T, N, torch::RestrictPtrTraits>;
using cutlass::gemm::GemmShape;

int near2power(int num)
{
    if (num <= 8)
        return 8;
    if (num <= 16)
        return 16;
    if (num <= 32)
        return 32;
    if (num <= 64)
        return 64;
    if (num <= 128)
        return 128;
    printf(" channel size of %d is too big\n", num);
    exit(-1);
    return 0;
}

namespace device
{

using cutlass::Status;

struct ConvBase {

    struct Arguments {
        torch::Tensor feature;    //  [NNZ, iC]
        torch::Tensor weight;     // [kernelVolume, iC, oC]
        torch::Tensor rules;      //  [NTile, kernelVolume, 2, NNZ ],
        torch::Tensor ruleSize;   // [Ntile, kernelVolume]
        torch::Tensor outFeature; // [outNNZ, oC]
        std::vector<int64_t> tile_grid_shape;

        Arguments() {}

        Arguments(
            const torch::Tensor& feature_,
            const torch::Tensor& weight_,
            const torch::Tensor& rules_,
            const torch::Tensor& ruleSize_,
            const torch::Tensor& outFeature_,
            const std::vector<int64_t>& tile_grid_shape)
            : feature(feature_),
              weight(weight_),
              rules(rules_),
              ruleSize(ruleSize_),
              outFeature(outFeature_),
              tile_grid_shape(tile_grid_shape)
        {
        }
    };

    virtual Status initialize(Arguments const& args) { return Status::kSuccess; };

    virtual Status run(hipStream_t stream = nullptr) { return Status::kSuccess; };

    virtual ~ConvBase() = default;
};

template <
    /// GemmShape, V, oC, iC
    typename ThreadBlockShape_,
    /// GemmShape, V, oC, iC
    typename WarpShape_,
    int VBLOCK,
    ///
    typename ThreadblockSwizzle_ =
        typename threadblock::IdentityThreadblockSwizzle>
struct Conv : public ConvBase {

    using ThreadblockSwizzle = ThreadblockSwizzle_;
    using ConvKernel = typename kernel::DefaultConv<
        ThreadBlockShape_, WarpShape_, VBLOCK, ThreadblockSwizzle_>::ConvKernel;

    static size_t get_workspace_size()
    {
        // TODO: use hipMemsetAsync(workspace, 0, bytes, stream)
        // might be useful when used in multiple streams
        return 0;
    }

private:
    typename ConvKernel::Params params_;
    int NTile_;

public:
    /// Constructs the Conv
    Conv()
    {
        // printf(" kWarpGemmIterations = %d \n", ConvKernel::Mma::kWarpGemmIterations);
    }

    Status initialize(Arguments const& args) override
    {
        NTile_ = args.ruleSize.size(0);
        int kernelVolume = args.weight.size(0);

        params_ = typename ConvKernel::Params(
            args.feature.template packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            args.weight.template packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
            args.rules.template packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
            args.ruleSize.template packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
            args.outFeature.template packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            kernelVolume);

        return Status::kSuccess;
    }

    Status run(hipStream_t stream = nullptr) override
    {
        ThreadblockSwizzle ts;

        dim3 grid = ts.get_grid_shape(NTile_);
        dim3 block(ConvKernel::kThreadCount, 1, 1);

        hipError_t result;

        int smem_size = int(sizeof(typename ConvKernel::SharedStorage));
        // printf("smem_size = %d\n", smem_size);

        if (smem_size >= (48 << 10)) {
            printf("info: use 48KB more SMEM\n");
            result = hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass::Kernel<ConvKernel>),
                                          hipFuncAttributeMaxDynamicSharedMemorySize,
                                          smem_size);

            if (result != hipSuccess) {
                printf(" error, hipFuncSetAttribute, dynam");
                return Status::kErrorInternal;
            }

            result = hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass::Kernel<ConvKernel>),
                                          hipFuncAttributePreferredSharedMemoryCarveout, 100);

            if (result != hipSuccess) {
                printf(" error, hipFuncSetAttribute, carveout");
                return Status::kErrorInternal;
            }
        }

        cutlass::Kernel<ConvKernel><<<grid, block, smem_size, stream>>>(params_);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // result = hipGetLastError();

        // return result == hipSuccess ? Status::kSuccess : Status::kErrorInternal;
        return Status::kSuccess;
    }
};

/**
* partial specilization for d_feature
*/
template <
    /// GemmShape, V, oC, iC
    typename ThreadBlockShape_,
    /// GemmShape, V, oC, iC
    typename WarpShape_,
    int VBLOCK>
struct Conv<ThreadBlockShape_, WarpShape_, VBLOCK, threadblock::InterleavedThreadblockSwizzle>
    : public ConvBase {

    using ThreadblockSwizzle = threadblock::InterleavedThreadblockSwizzle;
    using ConvKernel = typename kernel::DefaultConv<
        ThreadBlockShape_, WarpShape_, VBLOCK, ThreadblockSwizzle>::ConvKernel;

    static size_t get_workspace_size()
    {
        // TODO: use hipMemsetAsync(workspace, 0, bytes, stream)
        // might be useful when used in multiple streams
        return 0;
    }

private:
    typename ConvKernel::Params params_;
    int NTile_;
    int tile_grid_h_;
    int tile_grid_w_;

public:
    /// Constructs the Conv
    Conv()
    {
        // printf(" kWarpGemmIterations = %d \n", ConvKernel::Mma::kWarpGemmIterations);
    }

    Status initialize(Arguments const& args) override
    {
        NTile_ = args.ruleSize.size(0);
        tile_grid_h_ = args.tile_grid_shape[0];
        tile_grid_w_ = args.tile_grid_shape[1];
        int kernelVolume = args.weight.size(0);

        params_ = typename ConvKernel::Params(
            args.feature.template packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            args.weight.template packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
            args.rules.template packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
            args.ruleSize.template packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
            args.outFeature.template packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            kernelVolume, tile_grid_h_, tile_grid_w_);

        return Status::kSuccess;
    }

    Status run(hipStream_t stream = nullptr) override
    {
        ThreadblockSwizzle ts;

        dim3 grid = ts.get_grid_shape(NTile_, tile_grid_h_, tile_grid_w_);
        dim3 block(ConvKernel::kThreadCount, 1, 1);

        hipError_t result;

        int smem_size = int(sizeof(typename ConvKernel::SharedStorage));
        // printf("smem_size = %d\n", smem_size);

        if (smem_size >= (48 << 10)) {
            printf("info: use 48KB more SMEM\n");
            result = hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass::Kernel<ConvKernel>),
                                          hipFuncAttributeMaxDynamicSharedMemorySize,
                                          smem_size);

            if (result != hipSuccess) {
                printf(" error, hipFuncSetAttribute, dynam");
                return Status::kErrorInternal;
            }

            result = hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass::Kernel<ConvKernel>),
                                          hipFuncAttributePreferredSharedMemoryCarveout, 100);

            if (result != hipSuccess) {
                printf(" error, hipFuncSetAttribute, carveout");
                return Status::kErrorInternal;
            }
        }

        for (int i = 0; i < 9; i++) {
            params_.update_tile_idx(i);
            cutlass::Kernel<ConvKernel><<<grid, block, smem_size, stream>>>(params_);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }
        // result = hipGetLastError();
        // return result == hipSuccess ? Status::kSuccess : Status::kErrorInternal;
        return Status::kSuccess;
    }
};



torch::Tensor
rule_conv(const torch::Tensor feature,  // [NNZ, C]
          const torch::Tensor weight,   // [kernelVolume, iC, oC]
          const torch::Tensor rules,    // [NTile, kernelVolume, 2, nnz_max],
          const torch::Tensor ruleSize, // [Ntile, kernelVolume]
          int outNNZ)
{
    int iC = weight.size(1);
    int oC = weight.size(2);

    int IC_BLOCK = near2power(iC);
    int OC_BLOCK = near2power(oC);

    int NTile = ruleSize.size(0);

    // allocate outFeature ?
    torch::Tensor outFeature =
        torch::zeros({outNNZ, oC},
                     torch::dtype(feature.dtype()).device(feature.device()));

    std::shared_ptr<ConvBase> conv;

    switch (OC_BLOCK) {
    case 8:
        // if oc = 8
        // error: static assertion failed with "ThreadMap::Iterations::kColumn must be > 0"
        conv = std::make_shared<Conv<GemmShape<8, 32, 8>, GemmShape<8, 32, 8>, 8>>();
        break;
    case 16:
        // if oc = 16
        // error: static assertion failed with "ThreadMap::Iterations::kColumn must be > 0"
        conv = std::make_shared<Conv<GemmShape<8, 32, 8>, GemmShape<8, 32, 8>, 8>>();
        break;
    case 32:
        conv = std::make_shared<Conv<GemmShape<8, 32, 8>, GemmShape<8, 32, 8>, 8>>();
        break;
    case 64:
        conv = std::make_shared<Conv<GemmShape<8, 64, 8>, GemmShape<8, 32, 8>, 8>>();
        break;
    default:
        printf("unsupported oC = %d\n", oC);
    }

    ConvBase::Arguments args(feature, weight, rules, ruleSize, outFeature, {0,0});

    conv->initialize(args);

    conv->run();

    return outFeature;
}

torch::Tensor
rule_conv_d_feature(const torch::Tensor feature,  // [NNZ, C]
                    const torch::Tensor weight,   // [kernelVolume, iC, oC]
                    const torch::Tensor rules,    // [NTile, kernelVolume, 2, nnz_max],
                    const torch::Tensor ruleSize, // [Ntile, kernelVolume]
                    std::vector<int64_t> tile_grid_shape,
                    int outNNZ)
{
    CHECK_INPUT(feature);
    CHECK_INPUT(weight);
    CHECK_INPUT(rules);
    CHECK_INPUT(ruleSize);

    int iC = weight.size(1);
    int oC = weight.size(2);

    int IC_BLOCK = near2power(iC);
    int OC_BLOCK = near2power(oC);

    int NTile = ruleSize.size(0);

    // allocate outFeature ?
    torch::Tensor outFeature =
        torch::zeros({outNNZ, oC},
                     torch::dtype(feature.dtype()).device(feature.device()));

    std::shared_ptr<ConvBase> conv;

    switch (OC_BLOCK) {
    case 8:
        // if oc = 8
        // error: static assertion failed with "ThreadMap::Iterations::kColumn must be > 0"
        conv = std::make_shared<Conv<GemmShape<8, 32, 8>, GemmShape<8, 32, 8>, 8, threadblock::InterleavedThreadblockSwizzle>>();
        break;
    case 16:
        // if oc = 16
        // error: static assertion failed with "ThreadMap::Iterations::kColumn must be > 0"
        conv = std::make_shared<Conv<GemmShape<8, 32, 8>, GemmShape<8, 32, 8>, 8, threadblock::InterleavedThreadblockSwizzle>>();
        break;
    case 32:
        conv = std::make_shared<Conv<GemmShape<8, 32, 8>, GemmShape<8, 32, 8>, 8, threadblock::InterleavedThreadblockSwizzle>>();
        break;
    case 64:
        conv = std::make_shared<Conv<GemmShape<8, 64, 8>, GemmShape<8, 32, 8>, 8, threadblock::InterleavedThreadblockSwizzle>>();
        break;
    default:
        printf("unsupported oC = %d\n", oC);
    }

    ConvBase::Arguments args(feature, weight, rules, ruleSize, outFeature, tile_grid_shape);

    conv->initialize(args);

    conv->run();

    return outFeature;
}


std::vector<torch::Tensor>
rule_conv_backward(const torch::Tensor d_featureOut, // [outNNZ, oC]
                   const torch::Tensor feature,      // [NNZ, iC]
                   const torch::Tensor weight,       // [kernelVolume, iC, oC]
                   const torch::Tensor rules,        // [NTile, kernelVolume, 2, nnz_max],
                   const torch::Tensor ruleSize,     // [Ntile, kernelVolume]
                   std::vector<int64_t> tile_grid_shape)
{
    int kernelVolume = weight.size(0);
    int iC = weight.size(1);
    int oC = weight.size(2);

    int IC_BLOCK = near2power(iC);
    int OC_BLOCK = near2power(oC);

    int NNZ = feature.size(0);
    int NTile = ruleSize.size(0);

    // allocate d_feature
    // d_feature = d_featureOut * weight
    // TODO: weight [ic oc] to  [oc ic]
    torch::Tensor d_feature = rule_conv_d_feature(
        d_featureOut, weight, rules, ruleSize, tile_grid_shape, NNZ);

    torch::Tensor d_weight = torch::zeros(
        {kernelVolume, iC, oC}, torch::dtype(feature.dtype()).device(feature.device()));

    return {d_feature, d_weight};
}

} // namespace device

} // namespace sphconv
