#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "timer.h"
#include "debug_utils.h"

using namespace std;
using namespace torch::indexing;


namespace sphconv
{

template <typename T, int N>
using GpuTensor = torch::PackedTensorAccessor32<T, N, torch::RestrictPtrTraits>;

__host__ __device__ __forceinline__
int getInTileSize(int outTileSize, int stride, int kernelSize)
{
    assert(stride <= kernelSize);
    return (stride - 1) * outTileSize + kernelSize;
}

__host__ __device__ __forceinline__
int linearIdx(int x0, int x1, int x2, int x3, int D1, int D2, int D3) {
    return ((x0 * D1 + x1) * D2 + x2) * D3 + x3;
}

__host__ __device__ __forceinline__
int linearIdx(int x0, int x1, int x2, int D1, int D2) {
    return (x0 * D1 + x1) * D2 + x2;
}

__host__ __device__ __forceinline__
int linearIdx(int x0, int x1, int D1) {
    return x0 * D1 + x1;
}

template <typename IType>
__device__ __forceinline__
int getLocalShift(const GpuTensor<IType, 3> &zPtr, // [B, H, W]
                  int TileSizeH, int TileSizeW,
                  int H, int W, int base,
                  int b, int x, int y)
{
    // int tileIdxX = x / TileSizeH;
    int tileIdxY = y / TileSizeW;
    // int x0 = tileIdxX * TileSizeH;
    int y0 = tileIdxY * TileSizeW;

    // last element outside the tile,
    int a = ((b == 0 && x == 0 && y0 == 0) ?  0 : zPtr[b][x][y0 - 1]);

    return (-a + base);
}

__device__ __forceinline__
int getLinearTileIdx(int TileSize0, int TileSize1, int x, int y, int TileGridW)
{
    int tileIdxX = x / TileSize0;
    int tileIdxY = y / TileSize1;
    return linearIdx(tileIdxX, tileIdxY, TileGridW);
}

template <typename IType>
__device__ __forceinline__
int updateBase(const GpuTensor<IType, 3> &zPtr, int H, int W,
               int b, int x, int y, int TileSizeH, int TileSizeW)
{
    int tileIdxY = y / TileSizeW;
    int y0 = tileIdxY * TileSizeW;

    return zPtr[b][x][min(y0 + TileSizeW - 1, W - 1)] - ((b == 0 && x == 0 && y0 == 0) ? 0 : zPtr[b][x][y0 - 1]);
}


/**
 * @brief init the grid[B, H, W, D],
 *  grid is a mapping from spatial location to its target glaobal physical location
 *
 * fill grid with global indices
 *
 * TODO: fill local indices
 *
 * @return __global__
 */
template <typename IType>
__global__ void prepareSubMGridKernel(
    const GpuTensor<IType, 1> zIndices,
    const GpuTensor<IType, 3> zPtr, // TODO replace zPtr with exclusiveScan
    GpuTensor<IType, 4> grid,
    int B, int H, int W)
{
    IType x = threadIdx.x + blockDim.x * blockIdx.x;
    IType y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= H || y >= W)
        return;

    // for each voxel
    for (int b = 0; b < B; b++)
    {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : *(&zPtr[b][x][y] - 1);

        // diverge here, but we assume it's quick
        for (int pos = zStart; pos < zEnd; pos++)
        {
            IType z = zIndices[pos];
            grid[b][x][y][z] = pos;
        }
    }
}

/**
 * @brief init the grid[B, H, W, D],
 *  grid is a mapping from spatial location to its target global physical location
 *
 * fill grid with global indices,
 *
 *  we first fill it with 0,
 * fill output cell to ones
 * sum along  D
 *
 * TODO: fill local indices
 *
 * @return __global__
 */
template <typename IType>
__global__ void prepareGridKernel(
    const GpuTensor<IType, 1> zIndices,
    const GpuTensor<IType, 3> zPtr,
    GpuTensor<IType, 4> grid, // [B, oH, oW, oD]
    int B, int H, int W,
    int KH, int KW, int KD,
    int sH, int sW, int sD,
    int padH, int padW, int padD,
    int dH, int dW, int dD)
{
    IType oH = grid.size(1);
    IType oW = grid.size(2);
    IType oD = grid.size(3);

    IType x = threadIdx.x + blockDim.x * blockIdx.x;
    IType y = threadIdx.y + blockDim.y * blockIdx.y;
    IType k = threadIdx.z + blockDim.z * blockIdx.z;

    if (x >= H || y >= W)
        return;

    // (KH, KW, KD)
    // k =  kx * KH  ky kz
    IType k_H = k / (KW * KD);
    IType k_W = (k / KD) % KW;
    IType k_D = k % KD;

    IType oX = OutSpatial(k_H, x, sH, dH, padH);
    if (oX < 0 || oX >= oH)
        return;
    IType oY = OutSpatial(k_W, y, sW, dW, padW);
    if (oY < 0 || oY >= oW)
        return;

    /// for each input voxel, fill its output to 1
    for (int b = 0; b < B; b++)
    {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : *(&zPtr[b][x][y] - 1);

        // diverge here, but we assume it's quick
        for (int pos = zStart; pos < zEnd; pos++)
        {
            IType z = zIndices[pos];
            IType oZ = OutSpatial(k_D, z, sD, dD, padD);
            if (oZ < 0 || oZ >= oD)
                continue;

            grid[b][oX][oY][oZ] = IType(1);
        }
    }
}

/**
 *  create ozIndices
 *   -- recompute all oZ, like we did in prepareGridKernel
 *      is there a way to reuse info in 'grid' ?
 *
 *      YES: but we don't know if it would be faster or not.
 *      we can scan on grid,
 *      TODO: implement both and compare
 */
template <typename IType>
__global__ void getOzIndicesAndRulesKernel(
    const GpuTensor<IType, 1> zIndices, // [NNZ]
    GpuTensor<IType, 1> ozIndices,      // [NNZ']
    const GpuTensor<IType, 3> zPtr,     // [B, H, W]
    const GpuTensor<IType, 4> grid,
    GpuTensor<IType, 4> rules,    // [NTile, KKK, 4(2), DMax]
    GpuTensor<IType, 2> ruleSize, // number active index, [NTile, KKK]
    int B, int H, int W,
    int KH, int KW, int KD,
    int sH, int sW, int sD,
    int padH, int padW, int padD,
    int dH, int dW, int dD)
{
    IType oH = grid.size(1);
    IType oW = grid.size(2);
    IType oD = grid.size(3);

    IType x = threadIdx.x + blockDim.x * blockIdx.x;
    IType y = threadIdx.y + blockDim.y * blockIdx.y;
    IType k = threadIdx.z + blockDim.z * blockIdx.z;

    if (x >= H || y >= W)
        return;

    // (KH, KW, KD)
    // k =  kx * KH  ky kz
    IType k_H = k / (KW * KD);
    IType k_W = (k / KD) % KW;
    IType k_D = k % KD;

    IType oX = OutSpatial(k_H, x, sH, dH, padH);
    if (oX < 0 || oX >= oH)
        return;
    IType oY = OutSpatial(k_W, y, sW, dW, padW);
    if (oY < 0 || oY >= oW)
        return;

    IType nTile = 0; // TODO

    for (int b = 0; b < B; b++)
    {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : *(&zPtr[b][x][y] - 1);

        for (int pos = zStart; pos < zEnd; pos++)
        {
            IType z = zIndices[pos];
            IType oZ = OutSpatial(k_D, z, sD, dD, padD);

            if (oZ < 0 || oZ >= oD)
                continue;

            IType global_out_idx = grid[b][oX][oY][oZ] - 1;

            // printf("k_D, z  = %d, %d,(oX,oY,oZ) = %d,%d,%d iIdx = %d,  oIdx = %d\n", k_D, z, oX,oY,oZ, pos, global_out_idx);

            IType counter = atomicAdd(&ruleSize[nTile][k], IType(1));

            rules[nTile][k][0][counter] = pos;
            rules[nTile][k][1][counter] = global_out_idx;

            // this assigned for many times, with the same value
            ozIndices[global_out_idx] = oZ;
        }
    }
}

/***
 *  fill rules,
        rules: [NTile, K*K*K, 4, DMax]
 */
template <typename IType>
__global__ void getSubMRulesKernel(
    const GpuTensor<IType, 1> zIndices,
    const GpuTensor<IType, 3> zPtr,
    const GpuTensor<IType, 4> grid,
    GpuTensor<IType, 4> rules,
    GpuTensor<IType, 2> ruleSize, // number active index, [NTile, KKK]
    int B, int H, int W,
    int KH, int KW, int KD,
    int sH, int sW, int sD,
    int padH, int padW, int padD,
    int dH, int dW, int dD,
    int inTileSize0, int inTileSize1,
    int outTileSize0, int outTileSize1)
{
    // extern __shared__ int *shared;
    // IType *inTileGrid = &shared[0];
    // IType *outTileGrid = &shared[inTileGridSize];

    IType oH = grid.size(1);
    IType oW = grid.size(2);
    IType oD = grid.size(3);

    // IType x = threadIdx.x + blockDim.x * blockIdx.x;
    IType y = threadIdx.y + blockDim.y * blockIdx.y;
    IType k = threadIdx.z + blockDim.z * blockIdx.z;

    if (y >= W)
        return;

    // (KH, KW, KD)
    // k =  kx * KH  ky kz
    IType k_H = k / (KW * KD);
    IType k_W = (k / KD) % KW;
    IType k_D = k % KD;

    IType oY = OutSpatial(k_W, y, sW, dW, padW);
    if (oY < 0 || oY >= oW)
        return;

    // int TileGridH = divUp(oH, outTileSize0);
    int TileGridW = divUp(oW, outTileSize1);

    int baseIn = 0;
    int baseOut = 0;
    for (int b = 0; b < B; b++)
    {
        for (int x = 0; x < H; x++)
        {
            IType oX = OutSpatial(k_H, x, sH, dH, padH);
            if (oX < 0 || oX >= oH)
                continue;

            int zEnd = zPtr[b][x][y];
            int zStart = (b == 0 && x == 0 && y == 0) ? 0 : *(&zPtr[b][x][y] - 1);

            IType nTile = getLinearTileIdx(outTileSize0, outTileSize1, oX, oY, TileGridW);

            // diverge here
            for (int globalInIdx = zStart; globalInIdx < zEnd; globalInIdx++)
            {
                IType z = zIndices[globalInIdx];
                IType oZ = OutSpatial(k_D, z, sD, dD, padD);
                if (oZ < 0 || oZ >= oD)
                    continue;

                IType globalOutIdx = grid[b][oX][oY][oZ];
                if (globalOutIdx < 0)
                    continue;

                printf("nTile = %d\n", nTile);
                IType counter = atomicAdd(&ruleSize[nTile][k], IType(1));

                // grid[b][x][y][z] = pos;
                // rules: [NTile, K*K*K, 4, DMax]
                rules[nTile][k][0][counter] = globalInIdx;
                rules[nTile][k][1][counter] = globalOutIdx;
                // local input index
                rules[nTile][k][2][counter] = globalInIdx + getLocalShift(
                                                                zPtr, inTileSize0, inTileSize1,
                                                                H, W, baseIn, b, x, y);
                // local output index
                rules[nTile][k][3][counter] = globalOutIdx + getLocalShift(
                                                                 zPtr, outTileSize0, outTileSize1,
                                                                 H, W, baseOut, b, oX, oY);
            }
            // __syncthreads();
            baseIn = baseIn + updateBase(zPtr, H, W, b, x, y, inTileSize0, inTileSize1);
            baseOut = baseOut + updateBase(zPtr, oH, oW, b, oX, oY, outTileSize0, outTileSize1);
        } // x
    } // b
}


/**
 *  tile_size: tile_size is on the output feature map.
 *
 *
 * return rules
 *  rules: [NTile, K*K*K, 4, DMax]
 *
 * ROADMAP:
 * 1. only generate global indices
 * 2. generate both local indices and global indices
 *
 * ref:   getIndicePair
 */
std::vector<torch::Tensor>
get_rules_subm(torch::Tensor zIndices,               //  [NNZ]
                torch::Tensor zPtr,                   // [B, H, W]
                // torch::Tensor grid,                   // [B, H, W, D]
                int batchSize,
                std::vector<int64_t> spatialShape,    // H, W, D
                std::vector<int64_t> outSpatialShape, // H, W, D
                std::vector<int64_t> kernelSize,
                std::vector<int64_t> stride,
                std::vector<int64_t> padding,
                std::vector<int64_t> dilation)
{
    int H_BLOCK = 4;
    int W_BLOCK = 32;

    dim3 gridSize = dim3(divUp(spatialShape[0], H_BLOCK), divUp(spatialShape[1], W_BLOCK), 1);
    dim3 blockSize = dim3(H_BLOCK, W_BLOCK, 1);

    torch::Tensor grid = torch::full({batchSize, outSpatialShape[0], outSpatialShape[1], outSpatialShape[2]},
                                     /*value=*/-1, torch::dtype(torch::kInt32).device(zIndices.device()));

    prepareSubMGridKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        batchSize,
        spatialShape[0], spatialShape[1]);

    // printTensor<int>(grid, "grid", 0, 0, outSpatialShape[0], 0, outSpatialShape[1]);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    std::cout << "grid = " << grid << std::endl;

    int64_t kernelVolume = std::accumulate(kernelSize.begin(), kernelSize.end(), 1, std::multiplies<int64_t>());

    int outTileSize0 = 2; // TODO
    int outTileSize1 = 2;
    int NTile = divUp(outSpatialShape[0], outTileSize0) * divUp(outSpatialShape[1], outTileSize1);

    // allocate rules and indice Num
    torch::Tensor rules =
        torch::full({NTile, kernelVolume, 4, zIndices.size(0)},
                    /*value=*/-1, torch::dtype(torch::kInt32).device(zIndices.device()));
    // rules is allocated larger, to be trimed lalter

    torch::Tensor ruleSize =
        torch::zeros({NTile, kernelVolume}, torch::dtype(torch::kInt32).device(zIndices.device()));


    int inTileSize0 = getInTileSize(outTileSize0, stride[0], kernelSize[0]);
    int inTileSize1 = getInTileSize(outTileSize1, stride[1], kernelSize[1]);

    W_BLOCK = 8;
    gridSize = dim3(1, divUp(spatialShape[1], W_BLOCK), 1);
    blockSize = dim3(1, W_BLOCK, kernelVolume);
    // int inTileGridSize = batchSize * inTileSize0 * inTileSize1 * spatialShape[2];
    // int outTileGridSize = batchSize * outTileSize0 * outTileSize1 * outSpatialShape[2];
    // auto sharedMemorySize =  sizeof(int32_t) * (inTileGridSize +
    // outTileGridSize);
    getSubMRulesKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        rules.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        ruleSize.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        batchSize,
        spatialShape[0], spatialShape[1],
        kernelSize[0], kernelSize[1], kernelSize[2],
        stride[0], stride[1], stride[2],
        padding[0], padding[1], padding[2],
        dilation[0], dilation[1], dilation[2],
        inTileSize0, inTileSize1,
        outTileSize0, outTileSize1);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    // outZPtr
    // outIndices ?

    return {zIndices, zPtr, rules, ruleSize};
}


/**
 *  tile_size: tile_size is on the output feature map.
 *
 *
 * return rules
 *  rules: [NTile, K*K*K, 4, DMax]
 *
 * ROADMAP:
 * 1. only generate global indices
 * 2. generate both local indices and global indices
 *
 * ref:   getIndicePair
 */
std::vector<torch::Tensor>
get_rules(torch::Tensor zIndices, //  [NNZ]
          torch::Tensor zPtr,     // [B, H, W]
                                  //   torch::Tensor grid,     // [B, oH, oW, oD]
          int batchSize,
          std::vector<int64_t> spatialShape,    // H, W, D
          std::vector<int64_t> outSpatialShape, // oH, oW, oD
          std::vector<int64_t> kernelSize,
          std::vector<int64_t> stride,
          std::vector<int64_t> padding,
          std::vector<int64_t> dilation)
{
    const int H_BLOCK = 2;
    const int W_BLOCK = 16;

    int64_t kernelVolume = std::accumulate(kernelSize.begin(), kernelSize.end(), 1, std::multiplies<int64_t>());
    dim3 gridSize = dim3(divUp(spatialShape[0], H_BLOCK), divUp(spatialShape[1], W_BLOCK), 1);
    dim3 blockSize = dim3(H_BLOCK, W_BLOCK, kernelVolume);

    // printf(" befaore preapre geridf kernel a\n");
    // printf("launch config : (%d,%d,%d),(%d,%d,%d)\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);

    torch::Tensor grid = torch::zeros({batchSize, outSpatialShape[0], outSpatialShape[1], outSpatialShape[2]},
                                      torch::dtype(torch::kInt32).device(zIndices.device()));

    prepareGridKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        batchSize, spatialShape[0], spatialShape[1],
        kernelSize[0], kernelSize[1], kernelSize[2],
        stride[0], stride[1], stride[2],
        padding[0], padding[1], padding[2],
        dilation[0], dilation[1], dilation[2]);

    // printTensor<int>(grid, "grid", 0, 0, outSpatialShape[0], 0, outSpatialShape[1]);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    // std::cout << "grid(1) = " << grid << std::endl;

    grid = torch::cumsum(grid, 3, torch::kInt32); // [B, oH, oW, oD]

    // std::cout << "grid(2) = " << grid << std::endl;
    // here we want non inclusive scan, but pytorch only provides this.
    torch::Tensor ozPtr = torch::cumsum(grid.index({Slice(), Slice(), Slice(), -1}).reshape({-1}), 0, torch::kInt32)
                              .reshape({batchSize, outSpatialShape[0], outSpatialShape[1]});
    // [B, oH, oW]
    // PRINT_SHAPE(ozPtr);
    // PRINT_SHAPE(grid);
    // std::cout << "ozPtr = " << ozPtr << std::endl;
    torch::Tensor exclusiveScan = ozPtr.roll(1);
    exclusiveScan.index_put_({0, 0, 0}, 0);
    grid += exclusiveScan.unsqueeze(-1); // now grid is filled with global output index
    // std::cout << "grid(3) = " << grid << std::endl;

    int NTile = 1; // TODO, number of Tiles

    torch::Tensor rules = torch::full({NTile, kernelVolume, 2, zIndices.size(0)},
                                      /*value=*/-1, torch::dtype(torch::kInt32).device(zIndices.device()));
    // rules is allocated larger, to be trimed lalter
    // TODO, change 2 to 4, numAct
    // TODO, last dimension... is NNZ now, But not NNZ if NTile > 1
    // PRINT_SHAPE(rules);

    torch::Tensor ruleSize =
        torch::zeros({NTile, kernelVolume}, torch::dtype(torch::kInt32).device(zIndices.device()));
    // PRINT_SHAPE(ruleSize);

    int outNNZ = ozPtr.view({-1}).index({-1}).item<int>();
    torch::Tensor ozIndices = torch::empty({outNNZ}, torch::dtype(torch::kInt32).device(zIndices.device()));
    // PRINT_SHAPE(ozIndices);

    getOzIndicesAndRulesKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        ozIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        rules.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        ruleSize.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        batchSize, spatialShape[0], spatialShape[1],
        kernelSize[0], kernelSize[1], kernelSize[2],
        stride[0], stride[1], stride[2],
        padding[0], padding[1], padding[2],
        dilation[0], dilation[1], dilation[2]);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // outZPtr
    // outIndices ?

    return {ozIndices, ozPtr, rules, ruleSize};
}

} // namespace sphconv
