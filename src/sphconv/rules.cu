#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "timer.h"
#include "debug_utils.h"
#include "assert.h"

using namespace std;
using namespace torch::indexing;


namespace sphconv
{

template <typename T, int N>
using GpuTensor = torch::PackedTensorAccessor32<T, N, torch::RestrictPtrTraits>;

__host__ __device__ __forceinline__
int getInTileSize(int outTileSize, int stride, int kernelSize)
{
    assert(stride <= kernelSize);
    return stride * (outTileSize - 1) + kernelSize;
}

__host__ __device__ __forceinline__
int linearIdx(int x0, int x1, int x2, int x3, int D1, int D2, int D3) {
    return ((x0 * D1 + x1) * D2 + x2) * D3 + x3;
}

__host__ __device__ __forceinline__
int linearIdx(int x0, int x1, int x2, int D1, int D2) {
    return (x0 * D1 + x1) * D2 + x2;
}

__host__ __device__ __forceinline__
int linearIdx(int x0, int x1, int D1) {
    return x0 * D1 + x1;
}


__device__ __forceinline__
int getLinearTileIdx(int TileSize0, int TileSize1, int x, int y, int TileGridW)
{
    int tileIdxX = x / TileSize0;
    int tileIdxY = y / TileSize1;
    return linearIdx(tileIdxX, tileIdxY, TileGridW);
}


/**
 * @brief init the grid[B, H, W, D],
 *  grid is a mapping from spatial location to its target glaobal physical location
 *
 * fill grid with global indices
 *
 * TODO: fill local indices
 *
 * @return __global__
 */
template <typename IType>
__global__ void prepareSubMGridKernel(
    const GpuTensor<IType, 1> zIndices,
    const GpuTensor<IType, 3> zPtr, // TODO replace zPtr with exclusiveScan
    GpuTensor<IType, 4> grid,
    int B, int H, int W)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= H || y >= W)
        return;

    // for each voxel
    for (int b = 0; b < B; b++)
    {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : *(&zPtr[b][x][y] - 1);

        // diverge here, but we assume it's quick
        for (int pos = zStart; pos < zEnd; pos++)
        {
            int z = zIndices[pos];
            grid[b][x][y][z] = pos;
        }
    }
}

/**
 * @brief init the grid[B, oH, oW, oD],
 *  grid is a mapping from spatial location to its target global physical location
 *
 * fill grid with global indices,
 *
 *  we first fill it with 0,
 * fill output cell to ones
 * sum along  D
 *
 * TODO: fill local indices
 *
 * @return __global__
 */
template <typename IType>
__global__ void prepareGridKernel(
    const GpuTensor<IType, 1> zIndices,
    const GpuTensor<IType, 3> zPtr,
    GpuTensor<IType, 4> grid, // [B, oH, oW, oD]
    int B, int H, int W,
    int KH, int KW, int KD,
    int sH, int sW, int sD,
    int padH, int padW, int padD,
    int dH, int dW, int dD)
{
    int oH = grid.size(1);
    int oW = grid.size(2);
    int oD = grid.size(3);

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z;

    if (x >= H || y >= W)
        return;

    // (KH, KW, KD)
    // k =  kx * KH  ky kz
    int k_H = k / (KW * KD);
    int k_W = (k / KD) % KW;
    int k_D = k % KD;

    int oX = OutSpatial(k_H, x, sH, dH, padH);
    if (oX < 0 || oX >= oH)
        return;
    int oY = OutSpatial(k_W, y, sW, dW, padW);
    if (oY < 0 || oY >= oW)
        return;

    /// for each input voxel, fill its output to 1
    for (int b = 0; b < B; b++)
    {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : *(&zPtr[b][x][y] - 1);

        // diverge here, but we assume it's quick
        for (int pos = zStart; pos < zEnd; pos++)
        {
            int z = zIndices[pos];
            int oZ = OutSpatial(k_D, z, sD, dD, padD);
            if (oZ < 0 || oZ >= oD)
                continue;

            grid[b][oX][oY][oZ] = 1;
        }
    }
}

/**
 *  for std conv
 *  create ozIndices and Rule s.
 */
template <typename IType>
__global__ void getOzIndicesAndRulesKernel(
    const GpuTensor<IType, 1> zIndices, // [NNZ]
    GpuTensor<IType, 1> ozIndices,      // [NNZ']
    const GpuTensor<IType, 3> zPtr,     // [B, H, W]
    const GpuTensor<IType, 3> ozPtr,    // [B, oH, oW]
    const GpuTensor<IType, 4> grid,
    GpuTensor<IType, 4> rules,  // [NTile, KKK, 2, DMax]
    GpuTensor<IType, 2> ruleSize,    // number active index, [NTile, KKK]
    int B, int H, int W, int D,  // TODO, cleanup unnaccesary
    int KH, int KW, int KD,
    int sH, int sW, int sD,
    int padH, int padW, int padD,
    int dH, int dW, int dD,
    int inTileH, int inTileW,
    int outTileH, int outTileW)
{
    int oH = grid.size(1);
    int oW = grid.size(2);
    int oD = grid.size(3);

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;

    if (x >= H || y >= W)
        return;

    int k_H = k / (KW * KD);
    int k_W = (k / KD) % KW;
    int k_D = k % KD;

    int oX = OutSpatial(k_H, x, sH, dH, padH);
    if (oX < 0 || oX >= oH)
        return;
    int oY = OutSpatial(k_W, y, sW, dW, padW);
    if (oY < 0 || oY >= oW)
        return;

    int TileGridW = divUp(oW, outTileW);
    int nTile = getLinearTileIdx(outTileH, outTileW, oX, oY, TileGridW);

    for (int b = 0; b < B; b++) {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : zPtr[b][x][y - 1];

        for (int globalInIdx = zStart; globalInIdx < zEnd; globalInIdx++) {
            int z = zIndices[globalInIdx];
            int oZ = OutSpatial(k_D, z, sD, dD, padD);
            if (oZ < 0 || oZ >= oD)
                continue;

            int globalOutIdx = grid[b][oX][oY][oZ] - 1;
            int counter = atomicAdd(&ruleSize[nTile][k], 1);

            if (counter < TILE_N_MAX) {
                rules[nTile][k][0][counter] = globalInIdx;
                rules[nTile][k][1][counter] = globalOutIdx;
            } else {
                printf("overflow counter:(%d/%d), global i/o:%d/%d, nTile:%d, x:%d, y:%d, k:%d, Tile(%d,%d), inShape(%d,%d,%d), std\n",
                       counter, TILE_N_MAX, globalInIdx, globalOutIdx,
                       nTile, x, y, k, inTileH, inTileW, H, W, D);
            }
            ozIndices[globalOutIdx] = oZ;
        }
    } // b
}

/***
 *  fill rules,
        rules: [NTile, K*K*K, 4, DMax]
 */
template <typename IType>
__global__ void getSubMRulesKernel(
    const GpuTensor<IType, 1> zIndices,
    const GpuTensor<IType, 3> zPtr,
    const GpuTensor<IType, 4> grid,
    GpuTensor<IType, 4> rules,
    GpuTensor<IType, 2> ruleSize, // number active index, [NTile, KKK]
    int B, int H, int W, int D,
    int KH, int KW, int KD,
    int sH, int sW, int sD,
    int padH, int padW, int padD,
    int dH, int dW, int dD,
    int inTileH, int inTileW,
    int outTileH, int outTileW)
{
    int oH = grid.size(1);
    int oW = grid.size(2);
    int oD = grid.size(3);

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= H || y >= W)
        return;

    int k = threadIdx.z;
    int k_H = k / (KW * KD);
    int k_W = (k / KD) % KW;
    int k_D = k % KD;

    int oX = OutSpatial(k_H, x, sH, dH, padH); // TODO, iterative
    if (oX < 0 || oX >= oH)
        return;
    int oY = OutSpatial(k_W, y, sW, dW, padW);
    if (oY < 0 || oY >= oW)
        return;

    // int TileGridH = divUp(oH, outTileH);
    int TileGridW = divUp(oW, outTileW);
    int nTile = getLinearTileIdx(outTileH, outTileW, oX, oY, TileGridW);

    for (int b = 0; b < B; b++)
    {
        int zEnd = zPtr[b][x][y];
        int zStart = (b == 0 && x == 0 && y == 0) ? 0 : zPtr[b][x][y - 1];

        // diverge here
        for (int globalInIdx = zStart; globalInIdx < zEnd; globalInIdx++) {
            int z = zIndices[globalInIdx];
            int oZ = OutSpatial(k_D, z, sD, dD, padD);
            if (oZ < 0 || oZ >= oD)
                continue;

            int globalOutIdx = grid[b][oX][oY][oZ];
            if (globalOutIdx < 0)
                continue;

            int counter = atomicAdd(&ruleSize[nTile][k], 1);

            if (counter < TILE_N_MAX) {
                rules[nTile][k][0][counter] = globalInIdx;
                rules[nTile][k][1][counter] = globalOutIdx;
            } else {
                printf("overflow counter:(%d/%d), global i/o:%d/%d, nTile:%d, x:%d, y:%d, k:%d, Tile(%d,%d), inShape(%d,%d,%d), std\n",
                       counter, TILE_N_MAX, globalInIdx, globalOutIdx,
                       nTile, x, y, k, inTileH, inTileW, H, W, D);
            }
        }
    } // b
}


/**
 *  tile_size: tile_size is on the output feature map.
 *
 *
 * return rules
 *  rules: [NTile, K*K*K, 4, DMax]
 *
 * ROADMAP:
 * 1. only generate global indices
 * 2. generate both local indices and global indices
 *
 * ref:   getIndicePair
 */
std::vector<torch::Tensor>
get_rules_subm(torch::Tensor zIndices, //  [NNZ]
               torch::Tensor zPtr,     // [B, H, W]
               int batchSize,
               std::vector<int64_t> spatialShape,    // H, W, D
               std::vector<int64_t> outSpatialShape, // H, W, D
               std::vector<int64_t> kernelSize,
               std::vector<int64_t> stride,
               std::vector<int64_t> padding,
               std::vector<int64_t> dilation)
{

    torch::Tensor grid = torch::full({batchSize, outSpatialShape[0], outSpatialShape[1], outSpatialShape[2]},
                                     /*value=*/-1, torch::dtype(torch::kInt32).device(zIndices.device()));

    dim3 gridSize(divUp(spatialShape[0], 16), divUp(spatialShape[1], 32));
    dim3 blockSize(16, 32, 1);
    prepareSubMGridKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        batchSize,
        spatialShape[0], spatialShape[1]);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    int64_t kernelVolume = std::accumulate(kernelSize.begin(), kernelSize.end(), 1, std::multiplies<int64_t>());

    int outTileH = 6; // TODO
    int outTileW = 6;
    if (outSpatialShape[0] <= 8) {
        outTileH = 4;
    }

    int NTile = divUp(outSpatialShape[0], outTileH) * divUp(outSpatialShape[1], outTileW);

    // allocate rules and indice Num
    torch::Tensor rules =
        torch::full({NTile, kernelVolume, 2, TILE_N_MAX}, // TODO: TILE_N_MAX is fixed, not elegent
                    /*value=*/-1, torch::dtype(torch::kInt32).device(zIndices.device()));
    // TODO: rules is allocated larger, to be trimed lalter

    torch::Tensor ruleSize =
        torch::zeros({NTile, kernelVolume}, torch::dtype(torch::kInt32).device(zIndices.device()));

    int inTileH = getInTileSize(outTileH, stride[0], kernelSize[0]);
    int inTileW = getInTileSize(outTileW, stride[1], kernelSize[1]);

    gridSize = dim3(divUp(spatialShape[0], 4), divUp(spatialShape[1], 8), 1);
    blockSize = dim3(4, 8, kernelVolume);
    getSubMRulesKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        rules.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        ruleSize.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        batchSize,
        spatialShape[0], spatialShape[1], spatialShape[2],
        kernelSize[0], kernelSize[1], kernelSize[2],
        stride[0], stride[1], stride[2],
        padding[0], padding[1], padding[2],
        dilation[0], dilation[1], dilation[2],
        inTileH, inTileW,
        outTileH, outTileW);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return {zIndices, zPtr, rules, ruleSize};
}

/**
 *  tile_size: tile_size is on the output feature map.
 *
 *
 * return rules
 *  rules: [NTile, K*K*K, 4, DMax]
 *
 * ROADMAP:
 * 1. only generate global indices
 * 2. generate both local indices and global indices
 *
 * ref:   getIndicePair
 */
std::vector<torch::Tensor>
get_rules(torch::Tensor zIndices, //  [NNZ]
          torch::Tensor zPtr,     // [B, H, W]
          int batchSize,
          std::vector<int64_t> spatialShape,    // H, W, D
          std::vector<int64_t> outSpatialShape, // oH, oW, oD
          std::vector<int64_t> kernelSize,
          std::vector<int64_t> stride,
          std::vector<int64_t> padding,
          std::vector<int64_t> dilation)
{
    torch::Tensor grid = torch::zeros({batchSize, outSpatialShape[0], outSpatialShape[1], outSpatialShape[2]},
                                      torch::dtype(torch::kInt32).device(zIndices.device()));
    int64_t kernelVolume = std::accumulate(kernelSize.begin(), kernelSize.end(), 1, std::multiplies<int64_t>());

    dim3 gridSize(divUp(spatialShape[0], 2), divUp(spatialShape[1], 16), 1);
    dim3 blockSize(2, 16, kernelVolume);
    prepareGridKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        batchSize, spatialShape[0], spatialShape[1],
        kernelSize[0], kernelSize[1], kernelSize[2],
        stride[0], stride[1], stride[2],
        padding[0], padding[1], padding[2],
        dilation[0], dilation[1], dilation[2]);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    grid = torch::cumsum(grid, 3, torch::kInt32); // [B, oH, oW, oD]

    // std::cout << "grid(2) = " << grid << std::endl;
    // here we want non inclusive scan, but pytorch only provides this.
    torch::Tensor ozPtr = torch::cumsum(grid.index({Slice(), Slice(), Slice(), -1}).reshape({-1}), 0, torch::kInt32)
                              .reshape({batchSize, outSpatialShape[0], outSpatialShape[1]});
    // [B, oH, oW]
    torch::Tensor exclusiveScan = ozPtr.roll(1);
    exclusiveScan.index_put_({0, 0, 0}, 0);
    grid += exclusiveScan.unsqueeze(-1); // now grid is filled with global output index
    // std::cout << "grid(3) = " << grid << std::endl;

    int outTileH = 8; // TODO
    int outTileW = 8;
    int inTileH = getInTileSize(outTileH, stride[0], kernelSize[0]);
    int inTileW = getInTileSize(outTileW, stride[1], kernelSize[1]);

    int NTile = divUp(outSpatialShape[0], outTileH) * divUp(outSpatialShape[1], outTileW);

    // TODO: rules is allocated larger, to be trimed lalter
    torch::Tensor rules = torch::full({NTile, kernelVolume, 2, TILE_N_MAX}, // TODO: TILE_N_MAX is fixed
                                      /*value=*/-1, torch::dtype(torch::kInt32).device(zIndices.device()));

    torch::Tensor ruleSize =
        torch::zeros({NTile, kernelVolume}, torch::dtype(torch::kInt32).device(zIndices.device()));
    // PRINT_SHAPE(ruleSize);

    int outNNZ = ozPtr.view({-1}).index({-1}).item<int>();
    torch::Tensor ozIndices = torch::empty({outNNZ}, torch::dtype(torch::kInt32).device(zIndices.device()));
    // PRINT_SHAPE(ozIndices);

    gridSize = dim3(divUp(spatialShape[0], 2), divUp(spatialShape[1], 16), 1);
    blockSize = dim3(2, 16, kernelVolume);
    getOzIndicesAndRulesKernel<int32_t><<<gridSize, blockSize>>>(
        zIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        ozIndices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        zPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        ozPtr.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        grid.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        rules.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
        ruleSize.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        batchSize, spatialShape[0], spatialShape[1], spatialShape[2],
        kernelSize[0], kernelSize[1], kernelSize[2],
        stride[0], stride[1], stride[2],
        padding[0], padding[1], padding[2],
        dilation[0], dilation[1], dilation[2],
        inTileH, inTileW,
        outTileH, outTileW);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return {ozIndices, ozPtr, rules, ruleSize};
}

} // namespace sphconv
