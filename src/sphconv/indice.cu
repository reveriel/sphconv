#include "hip/hip_runtime.h"
#include <debug_utils.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include "indice.cu.h"
#include "timer.h"

using namespace std;

namespace sphconv {

const int H_BLOCK = 4, W_BLOCK = 8;

static inline void report_time(const char *func, const char *event, CudaContextTimer<> &timer)
{
  // printf("%s:%s  %.3f\n", func, event, timer.report() / 1000.0);
}

///////
// return {new_depth, new_thick, InRuleMap, OutRuleMap, NumIn};
///////
std::vector<torch::Tensor>
get_indice_pairs(torch::Tensor depth,
                 torch::Tensor thick,
                 int D,
                 int KD, int KH, int KW,
                 int sD, int sH, int sW,
                 int padD, int padH, int padW,
                 int dD, int dH, int dW,
                 int groups)
{
  auto timer = CudaContextTimer<>();
  // assume we want to have each block to calcultate T output elements
  // (T + k - 1)^* inpuut elements are neededd ,

  // tile size
  // the output tlie
  // constexpr int H_TILE = 16, W_TILE = 16;
  int N = depth.size(0);
  int T = depth.size(1);
  int H = depth.size(2);
  int W = depth.size(3);

  int oD, oH, oW;
  oD = std::floor((D + 2 * padD - dD * (KD - 1) - 1) / sD + 1);
  oH = std::floor((H + 2 * padH - dH * (KH - 1) - 1) / sH + 1);
  oW = std::floor((W + 2 * padW - dW * (KW - 1) - 1) / sW + 1);
  int oT_MAX = T * 27;

  auto kernel_volume = KD * KH * KW;

  dim3 grid_size, block_size;
  auto new_depth = torch::zeros(
      {N, oT_MAX, oH, oW}, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

  auto new_thick = torch::zeros(
      {N, oH, oW}, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

  // count number of valid input voxel at (b,k,x,y)
  auto NumIn = torch::zeros({N, kernel_volume, H, W},
                   torch::dtype(torch::kInt32).device(torch::kCUDA, 0));
  // the thickness of the valid input voxel
  auto InRuleMap = torch::full({N, kernel_volume, H, W, T},
    /*value=*/ -1, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));
  // the output thickness of the valid input voxel
  auto OutRuleMap = torch::full({N, kernel_volume, H, W, T},
    /*value=*/ -1, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

  //// create <del>hash</del>map
  // the final value of CompactMap, means
  // the output thick + 1 at output coordinate, (b, oX, oY, oZ)
  auto CompactMap = torch::full({N, oH, oW, oD}, 0,
                  torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

  report_time(__FUNCTION__, "init", timer);

  const int oH_BLOCK = 8, oW_BLOCK = 32;

  printTensor_int(depth, "depth", 0, 0, H, 0, W);
  printTensor_int(thick, "thick", 0, 0, H, 0, W);

  grid_size = dim3(divUp(H, H_BLOCK), divUp(W, W_BLOCK));
  block_size = dim3(H_BLOCK, W_BLOCK, kernel_volume);
  get_indice_pairs_kernel_1<int32_t><<<grid_size, block_size>>>( // <scalar_t, int32_t, H_TILE, W_TILE>
      depth.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      thick.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
      NumIn.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      InRuleMap.packed_accessor32<int32_t, 5, torch::RestrictPtrTraits>(),
      OutRuleMap.packed_accessor32<int32_t, 5, torch::RestrictPtrTraits>(),
      CompactMap.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      N,
      H, W,
      KD, KH, KW,
      sD, sH, sW,
      padD, padH, padW,
      dD, dH, dW,
      oD, oH, oW);

  gpuErrchk(hipPeekAtLastError());
gpuErrchk(hipDeviceSynchronize());

  report_time(__FUNCTION__, "phase1", timer);

  printTensor_int(NumIn, "NumIn", 0, 0, H, 0, W);
  printTensor_k_int(InRuleMap, "InRuleMap", 0, 0, H, 0, W);
  printTensor_k_int(OutRuleMap, "OutRuleMap", 0, 0, H, 0, W);

  grid_size = dim3(divUp(oH, oH_BLOCK), divUp(oW, oW_BLOCK), 1);
  block_size = dim3(oH_BLOCK, oW_BLOCK, 1);

  get_indice_pairs_kernel_2<int32_t><<<grid_size, block_size>>>(
    CompactMap.packed_accessor32<int32_t, 4, RestrictPtrTraits>(),
    new_depth.packed_accessor32<int32_t, 4, RestrictPtrTraits >(),
    new_thick.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
    N,
    kernel_volume,
    oH, oW, oD);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  report_time(__FUNCTION__, "phase2", timer);

  printTensor_int(new_depth, "new_depth", 0, 0, oH, 0, oW);
  printTensor_int(new_thick, "new_thick", 0, 0, oH, 0, oW);
  // std::cout << "CompactMap = " << CompactMap << std::endl;

  grid_size = dim3(divUp(H, H_BLOCK * 4), divUp(W, W_BLOCK * 4), 1);
  block_size = dim3(H_BLOCK * 4, W_BLOCK * 4, 1);

  get_indice_pairs_kernel_3<int32_t><<<grid_size, block_size>>>(
    CompactMap.packed_accessor32<int32_t, 4, RestrictPtrTraits>(),
    OutRuleMap.packed_accessor32<int32_t, 5, RestrictPtrTraits>(),
    NumIn.packed_accessor32<int32_t, 4, RestrictPtrTraits>(),
    N,
    H, W,
    kernel_volume,
    KD, KH, KW,
    sH, sW,
    padH, padW,
    dH, dW,
    oH, oW);

  report_time(__FUNCTION__, "phase3", timer);

  int oT = torch::max(new_thick).item<int>();
  // only supported in pytorch 1.5
  // new_depth = new_depth.index({Ellipsis, Slice(0, oT), Ellipsis, Ellipsis}).contiguous();
  new_depth = new_depth.narrow(1, 0, oT).contiguous();

  // printf(" oT / oT_MAX = %d / %d\n", oT, oT_MAX );
  // printf("thickness: iT = %d, oT = %d, fullness = %.3lf\n", T, oT, ((double)N * oT * oH * oW) / torch::sum(new_thick).item<int>());

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  report_time(__FUNCTION__, "phase4", timer);

  printTensor_k_int(OutRuleMap, "OutRuleMap after k3", 0, 0, H, 0, W);

  return {new_depth, new_thick, InRuleMap, OutRuleMap, NumIn};
}

///////
// return {depth, thick, InRuleMap, OutRuleMap, NumIn};
///////
std::vector<torch::Tensor>
get_indice_pairs_subm(torch::Tensor depth,
                      torch::Tensor thick,
                      int D,
                      int KD, int KH, int KW,
                      int sD, int sH, int sW,
                      int padD, int padH, int padW,
                      int dD, int dH, int dW,
                      int groups)
{
  auto timer = CudaContextTimer<>();

  int N = depth.size(0);
  int T = depth.size(1);
  int H = depth.size(2);
  int W = depth.size(3);
  // output sizes
  int oD, oH, oW;
  oD = D;
  oH = H;
  oW = W;

  auto kernel_volume = KD * KH * KW;

  dim3 grid_size, block_size;

  // count number of valid input voxel at (b,k,x,y)
  auto NumIn = torch::zeros({N, kernel_volume, H, W},
                   torch::dtype(torch::kInt32).device(torch::kCUDA, 0));
  // the thickness of the valid input voxel
  auto InRuleMap = torch::full({N, kernel_volume, H, W, T},
    /*value=*/ -1, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));
  // the output thickness of the valid input voxel
  auto OutRuleMap = torch::full({N, kernel_volume, H, W, T},
    /*value=*/ -1, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

  //// create <del>hash</del>map
  // the final value of CompactMap, means
  // the output thick + 1 at output coordinate, (b, oX, oY, oZ)
  auto CompactMap = torch::full({N, oH, oW, oD}, 0,
                  torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

  report_time(__FUNCTION__, "init", timer);

  grid_size = dim3(divUp(H, H_BLOCK * 4), divUp(W, W_BLOCK * 4), 1);
  block_size = dim3(H_BLOCK * 4, W_BLOCK * 4, 1);
  get_indice_pairs_subm_kernel_1<int32_t><<<grid_size, block_size>>>(
      depth.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      thick.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
      CompactMap.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      N, H, W);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  report_time(__FUNCTION__, "phase1", timer);

  grid_size = dim3(divUp(H, H_BLOCK), divUp(W, W_BLOCK));
  block_size = dim3(H_BLOCK, W_BLOCK, kernel_volume);
  get_indice_pairs_subm_kernel_2<int32_t><<<grid_size, block_size>>>(
      depth.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      thick.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
      NumIn.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      InRuleMap.packed_accessor32<int32_t, 5, torch::RestrictPtrTraits>(),
      OutRuleMap.packed_accessor32<int32_t, 5, torch::RestrictPtrTraits>(),
      CompactMap.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
      N,
      H, W,
      KD, KH, KW,
      sD, sH, sW,
      padD, padH, padW,
      dD, dH, dW,
      oD, oH, oW);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  report_time(__FUNCTION__, "phase2 ", timer);

  return {depth, thick, InRuleMap, OutRuleMap, NumIn};
}

}
